#include "hip/hip_runtime.h"
#include "model.h"

#include <hip/hip_fp16.h>

#include <cfloat>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define FULL_MASK 0xffffffff

#define CUDA_CHECK(x)                                                                                    \
	do {                                                                                                 \
		hipError_t err = x;                                                                             \
		if (err != hipSuccess) {                                                                        \
			fprintf(stderr, "CUDA error in %s at %s:%d: %s (%s=%d)\n", __FUNCTION__, __FILE__, __LINE__, \
			        hipGetErrorString(err), hipGetErrorName(err), err);                                \
			abort();                                                                                     \
		}                                                                                                \
	} while (0)

#define CUDA_CHECK2(x, msg)                                                                                    \
	do {                                                                                                 \
		hipError_t err = x;                                                                             \
		if (err != hipSuccess) {                                                                        \
			fprintf(stderr, "[%s] CUDA error in %s at %s:%d: %s (%s=%d)\n", msg.c_str(), __FUNCTION__, __FILE__, __LINE__, \
			        hipGetErrorString(err), hipGetErrorName(err), err);                                \
			abort();                                                                                     \
		}                                                                                                \
	} while (0)

static void* cuda_devicecopy(void* host, size_t size) {
	void* device = NULL;
	CUDA_CHECK(hipMalloc(&device, size));
	CUDA_CHECK(hipMemcpyAsync(device, host, size, hipMemcpyHostToDevice));
	return device;
}

static void* cuda_hostcopy(void* device, size_t size, std::string debug = "") {
  void* host = NULL;
  CUDA_CHECK2(hipHostMalloc(&host, size), debug);
  CUDA_CHECK2(hipMemcpy(host, device, size, hipMemcpyDeviceToHost), debug);
  return host;
}

[[maybe_unused]] static void* cuda_devicealloc(size_t size) {
	void* ptr = NULL;
	CUDA_CHECK(hipMalloc(&ptr, size));
	return ptr;
}

[[maybe_unused]] static void* cuda_hostalloc(size_t size) {
	void* ptr = NULL;
	CUDA_CHECK(hipHostAlloc(&ptr, size, 0));
	return ptr;
}

extern "C" void* upload_cuda(void* host, size_t size) {
	return cuda_devicecopy(host, size);
}

extern "C" void* download_cuda(void* device, size_t size, std::string debug) {
  return cuda_hostcopy(device, size, debug);
}

extern "C" void register_cuda_host(void* host, size_t size) {
  CUDA_CHECK(hipHostRegister(host, size, hipHostRegisterDefault));
}

extern "C" void free_cuda(void* device) {
  CUDA_CHECK(hipFree(device));
}

extern "C" void unregister_cuda_host(void* host) {
  CUDA_CHECK(hipHostUnregister(host));
}

static int warp_size = 0;
static int max_threads_per_block = 0;
static int WARP_SIZE = 32;

extern "C" void set_cuda_device(int device) {
  CUDA_CHECK(hipSetDevice(device));
  CUDA_CHECK(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize, device));
  CUDA_CHECK(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock, device));
}

__device__ 
inline float warp_reduce_sum(float val) {
  for (int offset = warpSize / 2; offset > 0; offset /= 2)
    val += __shfl_down_sync(FULL_MASK, val, offset);

  return val;
}

__device__ 
inline float warp_all_reduce_max(float val) {
	// Max reduction across a warp.
	// All threads will contain the max of all threads in the warp.
  for (int mask = warpSize/2; mask > 0; mask /= 2) {
    val = max(val, __shfl_xor_sync(FULL_MASK, val, mask));
  }
  return val;
}

__device__ 
inline float block_all_reduce_max(float val) {
	// Max reduction across a 1-D block implemented as double warp max reduction.
	// All threads will contain the max of all threads in the block.
	
	// Will hold results of all warps.
	// Capacity 32 since there can be at most 32 warps in a thread.
  __shared__ float shared[32];
  const int wid  = threadIdx.x / warpSize;
  const int lane = threadIdx.x % warpSize;

  val = warp_all_reduce_max(val);

  if (blockDim.x < warpSize) return val;
  if (lane == 0) shared[wid] = val;

  __syncthreads();

  if ( wid == 0 ) {
	  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : -FLT_MAX;
  }
  val = warp_all_reduce_max(val);
  if (lane == 0) shared[wid] = val;
  
  __syncthreads();
  
  return shared[0];
}

__device__ 
inline float warp_all_reduce_sum(float val) {
	// Sum reduction across a warp.
	// All threads will contain the sum of all threads in the warp.
  for (int mask = warpSize/2; mask > 0; mask /= 2) {
    val += __shfl_xor_sync(FULL_MASK, val, mask);
  }
  return val;
}

__device__ 
inline float block_all_reduce_sum(float val) {
	// Sum reduction across a 1-D block implemented as double warp sum reduction.
	// All threads will contain the sum of all threads in the block.
	
	// Will hold results of all warps.
	// Capacity 32 since there can be at most 32 warps in a thread.
  __shared__ float shared[32];
  const int wid  = threadIdx.x / warpSize;
  const int lane = threadIdx.x % warpSize;

  val = warp_all_reduce_sum(val);

  if (blockDim.x < warpSize) return val;
  if (lane == 0) shared[wid] = val;

  __syncthreads();

  if ( wid == 0 ) {
	  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0.0;
  }
  val = warp_all_reduce_sum(val);
  if (lane == 0) shared[wid] = val;
  
  __syncthreads();
  
  return shared[0];
}

__global__
void matmul(const float* A, const float* x, int n, int d, float* out) {
	// A (d,n) @ x (n,) -> out (d,)
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i >= d) return;
	float sum = 0.0;
	for(int j = 0; j < n; j++){
		sum += A[n * i + j] * x[j];
	} 
	out[i] = sum;
}

__global__
void matmul(const half* A, const float* x, int n, int d, float* out) {
		// A (d,n) @ x (n,) -> out (d,)
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i >= d) return;
	float sum = 0.0;
	for(int j = 0; j < n; j++){
		sum += __half2float(A[n * i + j]) * x[j];
	} 
	out[i] = sum;
}



__device__
inline float matmul_row(const float* row, const float* x, int offset, int dim) {
  float sum = 0.0;
  for (int j = offset; j < dim; j += 32) {
    float v = row[j] * x[j];
    sum += v;
  }
  return warp_reduce_sum(sum);
}

__device__
inline float matmul_row(const half* row, const float* x, int offset, int dim) {
	float sum = 0.0;
	for (int j = offset; j < dim; j += 32) {
		float v = __half2float(row[j]) * x[j];
		sum += v;
	}
	return warp_reduce_sum(sum);
}

template <typename T>
__global__
inline void matmul_full_utilization(const T* A, const float* x, int n, int d, float* out){
	// A (d,n) @ x (n,) -> out (d,)
	int i = blockIdx.x;
	if (i >= d) return;
	int offset = threadIdx.x;
	float rowSum = matmul_row(&A[n * i], x, offset, n);
	if (threadIdx.x == 0) {
    	out[i] = rowSum;
	}
}

template <typename T>
__global__
inline void fused_matmul_full_utilization(const T* A, const float* x, int n, int d, float* out){
	// A (d,n) @ x (n,) -> out (d,)
	int i = blockIdx.x;
	if (i >= d) return;
	int offset = threadIdx.x;
	float rowSum = matmul_row(&A[n * i], x, offset, n);
	if (threadIdx.x == 0) {
    	out[i] += rowSum;
	}
}


__device__ inline float blocktranspose(float v, float def) {
  // Performs block-and-warp transpose operation:
  //   For a block containing K warps where lane 0 contains val_k,
  //   this function returns:
  //   - For warp 0, lane K: val_k
  //   - For all other warps and lanes: def
  int lane = threadIdx.x % warpSize;
  int warp = threadIdx.x / warpSize;
  
  // Will hold results of all warps.
  // Each lane of the warp accumulates across 1 head element at a time.
  // NOTE: Assumes warpSize is 32
  __shared__ float sm[32];
  if (lane == 0) sm[warp] = v;
  __syncthreads();
  
  return lane < blockDim.x / warpSize ? sm[lane] : def;
}

template <typename T>
__global__
void matmul_wide(const T* A, const float* x, int n, int d, float* out) {
  // A (d,n) @ x (n,) -> out (d,)
  // PRECOND: Block is 1-D and contains WPB warps.
  int i = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
  if (i >= d) return;
  // Warp j computes sum for row at <blockIdx.x*WPB + j>
  // Lane 0 of each warp will hold result
  int k = threadIdx.x % warpSize; //k = offset
  float rowSum = matmul_row(&A[n * i], x, k, n);
  // Transpose values so lane k in warp 0 contains row at <blockIdx.x*WPB + k>
  // For WPB=32, this allows us to coalesce 32 float32 writes into a single 128-byte store
  rowSum = blocktranspose(rowSum, 1.0);
  if (threadIdx.x < blockDim.x / warpSize) {
    int block_start_i = blockIdx.x * blockDim.x / warpSize;
    out[block_start_i + k] = rowSum;
  }
}

template <typename T>
__global__
void fused_matmul_add_residuals(const T* A, const float* x, int n, int d, float* out) {
  // A (d,n) @ x (n,) -> out (d,)
  // PRECOND: Block is 1-D and contains WPB warps.
  int i = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
  if (i >= d) return;
  // Warp j computes sum for row at <blockIdx.x*WPB + j>
  // Lane 0 of each warp will hold result
  int k = threadIdx.x % warpSize;
  float rowSum = matmul_row(&A[n * i], x, k, n);
  // Transpose values so lane k in warp 0 contains row at <blockIdx.x*WPB + k>
  // For WPB=32, this allows us to coalesce 32 float32 writes into a single 128-byte store
  rowSum = blocktranspose(rowSum, 1.0);
  if (threadIdx.x < blockDim.x / warpSize) {
    int block_start_i = blockIdx.x * blockDim.x / warpSize;
    out[block_start_i + k] += rowSum;
  }
}

template <typename T>
inline void dispatch_matmul(const T* A, const float* x, int n, int d, float* out) {// deli: n is embedding dimension, d is head dimension, kv_head_dim, or others. 
	
	int BLOCK_SIZE = WARP_SIZE;
	matmul_full_utilization<<<d, BLOCK_SIZE>>>(A, x, n, d, out);
}

template <typename T>
inline void dispatch_fused_matmul(const T* A, const float* x, int n, int d, float* out) {// deli: n is embedding dimension, d is head dimension, kv_head_dim, or others. 
	int BLOCK_SIZE = WARP_SIZE;
	fused_matmul_add_residuals<<<d, BLOCK_SIZE>>>(A, x, n, d, out);
}



template <typename T>
__global__
void fused_qkv_matmul_clip(
	const T* wq,      // (q_dim, dim)
	const T* wk,      // (kv_dim, dim)
	const T* wv,      // (kv_dim, dim)
	const float* x,   // (dim,)
	int dim,          // input dimension
	int q_dim,        // n_heads * head_dim
	int kv_dim,       // n_kv_heads * head_dim
	float qkv_clip,   // clipping value
	float* q_out,     // (q_dim,)
	float* k_out,     // (kv_dim,)
	float* v_out      // (kv_dim,)
) {
	// Each warp handles one row of either Q, K, or V output
	int warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
	int total_rows = q_dim + 2 * kv_dim;
	if (warp_id >= total_rows) return;
	
	// Determine which matrix (Q, K, or V) we're computing
	const T* w;
	float* out;
	if (warp_id < q_dim) {
		// Computing Q
		w = wq + warp_id * dim;
		out = q_out + warp_id;
	} else if (warp_id < q_dim + kv_dim) {
		// Computing K
		w = wk + (warp_id - q_dim) * dim;
		out = k_out + (warp_id - q_dim);
	} else {
		// Computing V
		w = wv + (warp_id - q_dim - kv_dim) * dim;
		out = v_out + (warp_id - q_dim - kv_dim);
	}

	// Compute matrix multiplication for this row
	// Since block is 1-dimensional, thread ID is same as threadIdx.x,
	// and warp partitions thread IDs
	int offset = threadIdx.x % warpSize;
	float row_sum = matmul_row(w, x, offset, dim);
	// Write result with clipping
	if (offset == 0) {
		row_sum = row_sum < -qkv_clip ? -qkv_clip : (row_sum > qkv_clip ? qkv_clip : row_sum);
		*out = row_sum;
	}
}

__global__
void attn(
	const float* kb,  // (max_seq_len, n_kv_heads, head_dim) 
	const float* q,   // (n_heads, head_dim)
	int head_dim, 
	int kv_len, 
	int max_seq_len, 
	int n_heads, 
  	int n_kv_heads,
	float* out        // (n_heads, kv_len)
) {
	int group = blockIdx.y;
	int t = blockIdx.x * blockDim.x + threadIdx.x;
	int h = blockIdx.y * blockDim.y + threadIdx.y;
	if (t >= kv_len || h >= n_heads) return;
	
	const float* query = q + h * head_dim;
	const float* key = kb + n_kv_heads * head_dim * t + head_dim * group;
	float score = 0.0;
	for (int i = 0; i < head_dim; i++) {
		score += query[i] * key[i];
	}
  out[h * max_seq_len + t] = score / sqrtf((float)head_dim);
}

__global__
void attn_softmax(
	const float* att, 
	int seq_len, 
	int max_seq_len, 
	int n_heads, 
	float* out
) {
	int offset = threadIdx.x;
	int h = blockIdx.x;
	int block_size = blockDim.x;
	if (h >= n_heads) return;
	
	const float* atth = att + max_seq_len * h;
	float* outh = out + max_seq_len * h;
	
	float score_max = -FLT_MAX;
	for (int t = offset; t < seq_len; t += block_size) {
		if (atth[t] > score_max) {
			score_max = atth[t];
		}
	}
	score_max = block_all_reduce_max(score_max);
	float score_sum = 0.0f;
	for (int t = offset; t < seq_len; t += block_size) {
		outh[t] = expf(atth[t] - score_max);
		score_sum += outh[t];
	}
	score_sum = block_all_reduce_sum(score_sum);
	for (int t = offset; t < seq_len; t += block_size) {
		outh[t] /= score_sum;
	}
}


//naive att_mix
__global__
void att_mix(
  const float* vb,  // (max_seq_len, n_kv_heads, head_dim) 
  const float* att, // (n_heads, kv_len)
  int head_dim, 
  int n_heads, 
  int n_kv_heads,
  int seq_len, 
  int max_seq_len, 
  float* out // (n_heads, head_dim)
) {
  // PRECOND: blocks are 2-D (warp_size, t_stride)
  int h = blockIdx.x;
  int group_size = n_heads / n_kv_heads;
  int g = h / group_size;
  int kv_stride = n_kv_heads * head_dim;
  
  const float* atth = att + max_seq_len * h;
  const float* vh = vb + head_dim * g;
  float* outh = out + head_dim * h;
  
  int warp_id = threadIdx.y;
  int t_stride = blockDim.y;
  
  // Capacity 32 since there can be at most 32 warps in a block.
  __shared__ float shared[32];
  
  for (int i = threadIdx.x; i < head_dim; i += warpSize) {
    if (warp_id == 0) {
      shared[threadIdx.x] = 0;
    }
    __syncthreads();
    float sum = 0.0;
    for (int t = warp_id; t < seq_len; t += t_stride) {
      sum += vh[kv_stride * t + i] * atth[t];	
    }
    atomicAdd(&shared[threadIdx.x], sum);
    __syncthreads();
    if (warp_id == 0) {
      outh[i] = shared[threadIdx.x];
      shared[threadIdx.x] = 0;
    }
  }
}

template <ActivationType A> __device__ inline float act(float x);
template<> __device__ inline float act<ActivationType::SILU>(float x) {
  return x / (1.0f + expf(-x));
}
template<> __device__ inline float act<ActivationType::GELU>(float x) {
  float x3 = x * x * x;
  return 0.5f * x * (1.0f + tanhf(0.797885f * (x + 0.044715f * x3)));
}

template <typename T, ActivationType A>
__global__
void fused_ffn_w1_w3_glu_act(
  const T* w1,        // (hidden_dim, dim)
  const T* w3,        // (hidden_dim, dim)
  const float* x,     // (dim,)
  int dim,           
  int hidden_dim,
  float* out         // (hidden_dim,)
) {
  // Each warp computes one row of both w1(x) and w3(x), then applies GLU
  int warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
  if (warp_id >= hidden_dim) return;
  
  int offset = threadIdx.x % warpSize;
  
  // Compute w1(x) and w3(x) for this row
  float sum1 = matmul_row(&w1[dim * warp_id], x, offset, dim);
  float sum3 = matmul_row(&w3[dim * warp_id], x, offset, dim);
  
  // Apply activation and multiply
  if (offset == 0) {
    out[warp_id] = act<A>(sum1) * sum3;
  }
}

__global__
void rmsnorm(const float* x, const float* weight, int size, float eps, float* out) {
	// PRECOND: only one 1-D block is launched
	float rms = 0.0;
	int offset = threadIdx.x;
	for (int i = offset; i < size; i += blockDim.x) {
		rms += x[i] * x[i];
	}
	rms = block_all_reduce_sum(rms);
	rms = sqrtf(rms / size + eps);
	float scale = 1.0 / rms;
	for (int i = offset; i < size; i += blockDim.x) {
		out[i] = x[i] * scale * weight[i];
	}
}

__device__
inline void rope(
  const float* x, int pair_idx, int head_dim, int pos, float theta, int rotary_dim, float* out
) {
  int j_head = pair_idx % head_dim;
  if (j_head < head_dim - 1) {  // Ensure we have a pair of elements
    float freq = j_head >= rotary_dim ? 0.f : 1.0f / powf(theta, (float)j_head / (float)rotary_dim);
    float val = pos * freq;
    float fcr = cosf(val);
    float fci = sinf(val);
    
    float2 v01 = *((float2*)&x[pair_idx]);
    float2 result = make_float2(
      v01.x * fcr - v01.y * fci,
      v01.x * fci + v01.y * fcr
    );
    *((float2*)&out[pair_idx]) = result;
  }
}

__global__
void fused_rope_and_cache_update(
  const float* q,         // (n_heads * head_dim,)
  const float* k,         // (n_kv_heads * head_dim,)
  const float* v,         // (n_kv_heads * head_dim,)
  int head_dim,          
  int n_heads,
  int n_kv_heads,
  int pos,               // current position
  int kv_pos,           // position in KV cache
  float theta,          // RoPE theta parameter
  int rotary_dim,       // how many dimensions to rotate
  float* q_out,         // (n_heads * head_dim,)
  float* kb,            // (max_seq_len, n_kv_heads, head_dim)
  float* vb            // (max_seq_len, n_kv_heads, head_dim)
) {
  // Each thread handles two consecutive elements (for RoPE complex rotation)
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int pair_idx = tid * 2;
  
  // Handle Q matrix RoPE
  if (pair_idx < n_heads * head_dim) {
    rope(
      q, pair_idx, head_dim, pos, 
      theta, rotary_dim, q_out
    );
  }
  
  // Handle K matrix RoPE and cache update
  if (pair_idx < n_kv_heads * head_dim) {
    float* k_out = &kb[kv_pos * (n_kv_heads * head_dim)];
    rope(
      k, pair_idx, head_dim, pos, 
      theta, rotary_dim, k_out
    );
  }
  
  // Handle V cache update (no RoPE needed)
  if (pair_idx < n_kv_heads * head_dim) {
    int cache_idx = kv_pos * (n_kv_heads * head_dim) + pair_idx;
    if (pair_idx < n_kv_heads * head_dim - 1) {
      vb[cache_idx] = (v[pair_idx]);
      vb[cache_idx + 1] = (v[pair_idx + 1]);
    }
  }
}

__global__
void rotate_sink_tokens(
  float* kb, 
  int kv_sink, 				// number of attention sinks
  int kv_dim, 				// size of each entry (all concatenated heads) in KV cache
  int head_dim,
  float theta, 				// RoPE theta parameter
  int rotary_dim			// how many dimensions to rotate
) {
  // Each thread handles two consecutive elements (for RoPE complex rotation)
  // across all attention sinks
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int pair_idx = tid * 2;
  
  if (pair_idx < kv_dim) {
    for (int r = 0; r < kv_sink; r++) {
      float* k = kb + r * kv_dim;
      rope(k, pair_idx, head_dim, 1, theta, rotary_dim, k);
    }
  }
}


__global__
void add_residuals(
	const float* x, const float* y, int d, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= d) return;
	
	out[i] = x[i] + y[i];
}

__global__
void clip(
	const float* x, float v, int d, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= d) return;
	
	out[i] = x[i] < -v ? -v : (x[i] > v ? v : x[i]);
}

__global__
void glu_silu(
	const float* x, const float* weight, int d, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= d) return;
	
	out[i] = weight[i] * x[i] / (1.0f + expf(-x[i]));
}

__global__
void glu_gelu(
	const float* x, const float* weight, int d, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= d) return;
	
	float v = x[i];
	out[i] =
		weight[i] * 
		0.5f * v * (1.0f + tanhf(0.797885f * (v + 0.044715f * v * v * v)));
}

// TODO: consolidate copy_embedding and copy_kv_entry into 1 memcpy kernel
__global__
void copy_embedding(
	const float* token_embedding_table, int dim, int token, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= dim) return;
	
	const float* v = token_embedding_table + dim * token;
	out[i] = v[i];
}

__global__
void copy_embedding(
	const half* token_embedding_table, int dim, int token, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= dim) return;
	
	const half* v = token_embedding_table + dim * token;
	out[i] = __half2float(v[i]);
}

__global__
void copy_kv_entry(
	const float* in, int kv_pos, int kv_dim, float* kb
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= kv_dim) return;
	
	kb[kv_pos * kv_dim + i] = in[i];
}


template <typename T>
void Block::_block_cuda(
	InferenceState& s, int pos, int kv_sink, int kv_pos, int kv_len
) const {
	const Config& c = *_config;
	
	// attention pre-norm
	switch (c.norm_type) {
    case LayerNormType::RMSNorm: {
	    rmsnorm<<<1, max_threads_per_block>>>(
				s.x(), rms_att_weight(), c.dim, c.norm_eps, s.xb()
			);
      break;
    }
  }
  
	int q_dim = c.n_heads * c.head_dim;
	int kv_dim = c.n_kv_heads * c.head_dim;

	// qkv matmuls for this position
	//dispatch_matmul<T>(wq<T>(), s.xb(), c.dim, q_dim, s.q());
	//dispatch_matmul<T>(wk<T>(), s.xb(), c.dim, kv_dim, s.k());
	//dispatch_matmul<T>(wv<T>(), s.xb(), c.dim, kv_dim, s.v());


	{
	// qkv matmuls for this position
	// some models require clipping qkv values
	int total_rows = q_dim + 2 * kv_dim;  // Total rows across Q, K, V
	fused_qkv_matmul_clip<<<total_rows, warp_size>>>(
		wq<T>(),
		wk<T>(),
		wv<T>(),
		s.xb(),
		c.dim,
		q_dim,
		kv_dim,
		c.qkv_clip,
		s.q(),
		s.k(),
		s.v()
	);
	}
  
	// key and value point to the kv cache
	float* kb = key_cache();
	float* vb = value_cache();

	//fused rope
	{
		// Calculate number of thread blocks needed
		// We need enough threads to handle the largest of:
		// - n_heads * head_dim (for Q)
		// - n_kv_heads * head_dim (for K and V)
		int max_dim = max(c.n_heads * c.head_dim, c.n_kv_heads * c.head_dim);
		int threads_needed = (max_dim + 1) / 2;  // Each thread handles 2 elements
		int num_blocks = (threads_needed + max_threads_per_block - 1) / max_threads_per_block;
		
		fused_rope_and_cache_update<<<num_blocks, max_threads_per_block>>>(
		s.q(),
		s.k(),
		s.v(),
		c.head_dim,
		c.n_heads,
		c.n_kv_heads,
		pos,
		kv_pos,
		c.rope_theta,
		c.rotary_dim,
		s.q(),           // Q can be updated in-place
		kb,
		vb
		);
	}
	
	if (kv_sink > 0) {
    // Sink tokens remain untouched while the rest of the KV cache is incrementally 
    // replaced in ring order, but sink i must always be positioned (max_seq_len - i)
    // away from current timestep. Hence, each forward pass, rotate sink tokens 
    // forward by 1. See https://arxiv.org/abs/2309.17453 for more.
    int threads_needed = (kv_dim + 1) / 2;  // Each thread handles 2 elements
    int num_blocks = (threads_needed + max_threads_per_block - 1) / max_threads_per_block;
    rotate_sink_tokens<<<num_blocks, max_threads_per_block>>>(
      kb, kv_sink, kv_dim, c.head_dim, c.rope_theta, c.rotary_dim
    );
  }
  

	// multihead attention: dot products and softmax
	{
		dim3 tpb;
		tpb.x = warp_size;
		tpb.y = c.n_heads / c.n_kv_heads;
		dim3 blocks;
		blocks.x = (kv_len + tpb.x - 1) / tpb.x;
		blocks.y = (c.n_heads + tpb.y - 1) / tpb.y;
		attn<<<blocks, tpb>>>(
			kb, s.q(), c.head_dim, kv_len, c.max_seq_len, c.n_heads, c.n_kv_heads, s.att()
		);
		attn_softmax<<<c.n_heads, warp_size>>>(
			s.att(), kv_len, c.max_seq_len, c.n_heads, s.att()
		);
	}
	
  // naive multihead attention: mix values with attention scores
	{
		dim3 tpb;
		tpb.x = warp_size;
		tpb.y = min(kv_len, max_threads_per_block / warp_size);
		dim3 blocks;
		blocks.x = c.n_heads;
		att_mix<<<blocks, tpb>>>(
		vb, s.att(),
		c.head_dim, c.n_heads, c.n_kv_heads, 
		kv_len, c.max_seq_len, s.xb2()
		);
	}
	
	// final matmul projection via wo, using `hb` as temp storage
	//dispatch_matmul<T>(wo<T>(), s.xb2(), q_dim, c.dim, s.hb());
	//dispatch_fused_matmul<T>(wo<T>(), s.xb2(), q_dim, c.dim, s.x());
	fused_matmul_add_residuals<<<c.dim/32, warp_size*32>>>(
    wo<T>(), s.xb2(), q_dim, c.dim, s.x()
  );
	
	// ffn pre-norm
	switch (c.norm_type) {
    case LayerNormType::RMSNorm: {
	    rmsnorm<<<1, max_threads_per_block>>>(
				s.x(), rms_ffn_weight(), c.dim, c.norm_eps, s.xb()
			);
      break;
    }
  }
	
	// mix self.w2(F.silu(self.w1(x)) * self.w3(x))
  // Note this is a feedforward with a GLU, not a simple MLP.
	switch (c.act) {
		case ActivationType::GELU: {
			fused_ffn_w1_w3_glu_act<T, ActivationType::GELU><<<
			c.hidden_dim, warp_size
		>>>(
			w1<T>(), w3<T>(), s.xb(), c.dim, c.hidden_dim, s.hb()
		);
			break;
		}
		case ActivationType::SILU: {
			fused_ffn_w1_w3_glu_act<T, ActivationType::SILU><<<
			c.hidden_dim, warp_size
		>>>(
			w1<T>(), w3<T>(), s.xb(), c.dim, c.hidden_dim, s.hb()
		);
			break;
		}
	}
	// add residual back: x <- w2(...) + x
	fused_matmul_add_residuals<<<c.dim/32, warp_size*32>>>(
		w2<T>(), s.hb(), c.hidden_dim, c.dim, s.x()
	);
    //dispatch_fused_matmul<T>(w2<T>(), s.hb(), c.hidden_dim, c.dim, s.x());
  	//dispatch_matmul<T>(w2<T>(), s.hb(), c.hidden_dim, c.dim, s.xb2());
	/*
	// ffn residual back into x
	add_residuals<<<
		(c.dim + max_threads_per_block - 1)/max_threads_per_block,
		max_threads_per_block
	>>>(
		s.x(), s.xb2(), c.dim, s.x()
	);
	*/
}

template void Block::_block_cuda<float>(InferenceState&, int, int, int, int) const;
template void Block::_block_cuda<half>(InferenceState&, int, int, int, int) const;
template<> void Block::_block_cuda<f16_t>(InferenceState& s, int pos, int kv_sink, int kv_pos, int kv_len) const {
  _block_cuda<half>(s, pos, kv_sink, kv_pos, kv_len);
}

void Model::_forward_cuda(InferenceState& s, int token, int pos, InferenceMode mode) {
	const Config& c = *config;
	
  switch (c.weight_dtype) {
    case DType::F32: {
	    copy_embedding<<<
        (c.dim + max_threads_per_block - 1)/max_threads_per_block,
        max_threads_per_block
      >>>(
        static_cast<float*>(token_embedding_table), c.dim, token, s.x()
      );
      break;
    }
    case DType::F16: {
	    copy_embedding<<<
        (c.dim + max_threads_per_block - 1)/max_threads_per_block,
        max_threads_per_block
      >>>(
        static_cast<half*>(token_embedding_table), c.dim, token, s.x()
      );
      break;
    }
    default: {
      assert(false && "unsupported weight dtype for CUDA");
    }
  }
	
	// When decoding past the context length, keep the first few tokens in the KV cache
	// untouched as "attention sinks" while replacing the rest in ring order.
	// See StreamingLLM (https://arxiv.org/pdf/2309.17453) for more.
	int kv_sink = pos >= c.max_seq_len ? KV_SINKS : 0;
	int kv_pos = kv_sink + (pos - kv_sink) % (c.max_seq_len - kv_sink);
	int kv_len = pos >= c.max_seq_len ? c.max_seq_len : pos + 1;
		
	// forward all layers in order
	for (auto b : blocks) {
		b->block(s, pos, kv_sink, kv_pos, kv_len);
	}

  if (mode == InferenceMode::HYDRATE_KV_CACHE) {
    // only hydrate the KV cache and don't compute output logits
	  CUDA_CHECK(hipGetLastError()); // check for kernel launch errors
    return;
  }
	
	// final layer norm
	switch (c.norm_type) {
		case LayerNormType::RMSNorm: {
			rmsnorm<<<1, max_threads_per_block>>>(
				s.x(), rms_final_weight, c.dim, c.norm_eps, s.x()
			);
			break;
		}
	}
	
	// classifier into logits
	switch (c.weight_dtype) {
    case DType::F32: {
	    dispatch_matmul<float>(
        static_cast<float*>(wcls), s.x(), c.dim, c.vocab_size, s.logits()
      );
      break;
    }
    case DType::F16: {
	    dispatch_matmul<half>(
        static_cast<half*>(wcls), s.x(), c.dim, c.vocab_size, s.logits()
      );
      break;
    }
    default: {
      assert(false && "unsupported weight dtype for CUDA");
    }
  }
	
	CUDA_CHECK(hipDeviceSynchronize()); // After this, s.logits contains logits of output token
	CUDA_CHECK(hipGetLastError()); // check for kernel launch errors
}