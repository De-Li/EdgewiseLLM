#include "hip/hip_runtime.h"
#include "model.h"

#include <hip/hip_fp16.h>

#include <cfloat>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define FULL_MASK 0xffffffff

#define CUDA_CHECK(x)                                                                                    \
	do {                                                                                                 \
		hipError_t err = x;                                                                             \
		if (err != hipSuccess) {                                                                        \
			fprintf(stderr, "CUDA error in %s at %s:%d: %s (%s=%d)\n", __FUNCTION__, __FILE__, __LINE__, \
			        hipGetErrorString(err), hipGetErrorName(err), err);                                \
			abort();                                                                                     \
		}                                                                                                \
	} while (0)

#define CUDA_CHECK2(x, msg)                                                                                    \
	do {                                                                                                 \
		hipError_t err = x;                                                                             \
		if (err != hipSuccess) {                                                                        \
			fprintf(stderr, "[%s] CUDA error in %s at %s:%d: %s (%s=%d)\n", msg.c_str(), __FUNCTION__, __FILE__, __LINE__, \
			        hipGetErrorString(err), hipGetErrorName(err), err);                                \
			abort();                                                                                     \
		}                                                                                                \
	} while (0)

static void* cuda_devicecopy(void* host, size_t size) {
	void* device = NULL;
	CUDA_CHECK(hipMalloc(&device, size));
	CUDA_CHECK(hipMemcpyAsync(device, host, size, hipMemcpyHostToDevice));
	return device;
}

static void* cuda_hostcopy(void* device, size_t size, std::string debug = "") {
  void* host = NULL;
  CUDA_CHECK2(hipHostMalloc(&host, size), debug);
  CUDA_CHECK2(hipMemcpy(host, device, size, hipMemcpyDeviceToHost), debug);
  return host;
}

[[maybe_unused]] static void* cuda_devicealloc(size_t size) {
	void* ptr = NULL;
	CUDA_CHECK(hipMalloc(&ptr, size));
	return ptr;
}

[[maybe_unused]] static void* cuda_hostalloc(size_t size) {
	void* ptr = NULL;
	CUDA_CHECK(hipHostAlloc(&ptr, size, 0));
	return ptr;
}

extern "C" void* upload_cuda(void* host, size_t size) {
	return cuda_devicecopy(host, size);
}

extern "C" void* download_cuda(void* device, size_t size, std::string debug) {
  return cuda_hostcopy(device, size, debug);
}

extern "C" void register_cuda_host(void* host, size_t size) {
  CUDA_CHECK(hipHostRegister(host, size, hipHostRegisterDefault));
}

extern "C" void free_cuda(void* device) {
  CUDA_CHECK(hipFree(device));
}

extern "C" void unregister_cuda_host(void* host) {
  CUDA_CHECK(hipHostUnregister(host));
}

static int warp_size = 0;
static int max_threads_per_block = 0;
static int WARP_SIZE = 32;

extern "C" void set_cuda_device(int device) {
  CUDA_CHECK(hipSetDevice(device));
  CUDA_CHECK(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize, device));
  CUDA_CHECK(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock, device));
}

__device__ 
inline float warp_reduce_sum(float val) {
  for (int offset = warpSize / 2; offset > 0; offset /= 2)
    val += __shfl_down_sync(FULL_MASK, val, offset);

  return val;
}

__device__ 
inline float warp_all_reduce_max(float val) {
	// Max reduction across a warp.
	// All threads will contain the max of all threads in the warp.
  for (int mask = warpSize/2; mask > 0; mask /= 2) {
    val = max(val, __shfl_xor_sync(FULL_MASK, val, mask));
  }
  return val;
}

__device__ 
inline float block_all_reduce_max(float val) {
	// Max reduction across a 1-D block implemented as double warp max reduction.
	// All threads will contain the max of all threads in the block.
	
	// Will hold results of all warps.
	// Capacity 32 since there can be at most 32 warps in a thread.
  __shared__ float shared[32];
  const int wid  = threadIdx.x / warpSize;
  const int lane = threadIdx.x % warpSize;

  val = warp_all_reduce_max(val);

  if (blockDim.x < warpSize) return val;
  if (lane == 0) shared[wid] = val;

  __syncthreads();

  if ( wid == 0 ) {
	  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : -FLT_MAX;
  }
  val = warp_all_reduce_max(val);
  if (lane == 0) shared[wid] = val;
  
  __syncthreads();
  
  return shared[0];
}

__device__ 
inline float warp_all_reduce_sum(float val) {
	// Sum reduction across a warp.
	// All threads will contain the sum of all threads in the warp.
  for (int mask = warpSize/2; mask > 0; mask /= 2) {
    val += __shfl_xor_sync(FULL_MASK, val, mask);
  }
  return val;
}

__device__ 
inline float block_all_reduce_sum(float val) {
	// Sum reduction across a 1-D block implemented as double warp sum reduction.
	// All threads will contain the sum of all threads in the block.
	
	// Will hold results of all warps.
	// Capacity 32 since there can be at most 32 warps in a thread.
  __shared__ float shared[32];
  const int wid  = threadIdx.x / warpSize;
  const int lane = threadIdx.x % warpSize;

  val = warp_all_reduce_sum(val);

  if (blockDim.x < warpSize) return val;
  if (lane == 0) shared[wid] = val;

  __syncthreads();

  if ( wid == 0 ) {
	  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0.0;
  }
  val = warp_all_reduce_sum(val);
  if (lane == 0) shared[wid] = val;
  
  __syncthreads();
  
  return shared[0];
}

__global__
void matmul(const float* A, const float* x, int n, int d, float* out) {
	// A (d,n) @ x (n,) -> out (d,)
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i >= d) return;
	float sum = 0.0;
	for(int j = 0; j < n; j++){
		sum += A[n * i + j] * x[j];
	} 
	out[i] = sum;
}

__global__
void matmul(const half* A, const float* x, int n, int d, float* out) {
		// A (d,n) @ x (n,) -> out (d,)
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i >= d) return;
	float sum = 0.0;
	for(int j = 0; j < n; j++){
		sum += __half2float(A[n * i + j]) * x[j];
	} 
	out[i] = sum;
}



__device__
inline float matmul_row(const float* row, const float* x, int offset, int dim) {
  float sum = 0.0;
  for (int j = offset; j < dim; j += 32) {
    float v = row[j] * x[j];
    sum += v;
  }
  return warp_reduce_sum(sum);
}

__device__
inline float matmul_row(const half* row, const float* x, int offset, int dim) {
	float sum = 0.0;
	for (int j = offset; j < dim; j += 32) {
		float v = __half2float(row[j]) * x[j];
		sum += v;
	}
	return warp_reduce_sum(sum);
}

template <typename T>
__global__
inline void matmul_full_utilization(const T* A, const float* x, int n, int d, float* out){
	// A (d,n) @ x (n,) -> out (d,)
	int i = blockIdx.x;
	if (i >= d) return;
	int offset = threadIdx.x;
	float rowSum = matmul_row(&A[n * i], x, offset, n);
	if (threadIdx.x == 0) {
    	out[i] = rowSum;
	}
}

template <typename T>
__global__
inline void fused_matmul_full_utilization(const T* A, const float* x, int n, int d, float* out){
	// A (d,n) @ x (n,) -> out (d,)
	int i = blockIdx.x;
	if (i >= d) return;
	int offset = threadIdx.x;
	float rowSum = matmul_row(&A[n * i], x, offset, n);
	if (threadIdx.x == 0) {
    	out[i] += rowSum;
	}
}


__device__ inline float blocktranspose(float v, float def) {
  // Performs block-and-warp transpose operation:
  //   For a block containing K warps where lane 0 contains val_k,
  //   this function returns:
  //   - For warp 0, lane K: val_k
  //   - For all other warps and lanes: def
  int lane = threadIdx.x % warpSize;
  int warp = threadIdx.x / warpSize;
  
  // Will hold results of all warps.
  // Each lane of the warp accumulates across 1 head element at a time.
  // NOTE: Assumes warpSize is 32
  __shared__ float sm[32];
  if (lane == 0) sm[warp] = v;
  __syncthreads();
  
  return lane < blockDim.x / warpSize ? sm[lane] : def;
}

template <typename T>
__global__
void matmul_wide(const T* A, const float* x, int n, int d, float* out) {
  // A (d,n) @ x (n,) -> out (d,)
  // PRECOND: Block is 1-D and contains WPB warps.
  int i = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
  if (i >= d) return;
  // Warp j computes sum for row at <blockIdx.x*WPB + j>
  // Lane 0 of each warp will hold result
  int k = threadIdx.x % warpSize; //k = offset
  float rowSum = matmul_row(&A[n * i], x, k, n);
  // Transpose values so lane k in warp 0 contains row at <blockIdx.x*WPB + k>
  // For WPB=32, this allows us to coalesce 32 float32 writes into a single 128-byte store
  rowSum = blocktranspose(rowSum, 1.0);
  if (threadIdx.x < blockDim.x / warpSize) {
    int block_start_i = blockIdx.x * blockDim.x / warpSize;
    out[block_start_i + k] = rowSum;
  }
}

template <typename T>
inline void dispatch_matmul(const T* A, const float* x, int n, int d, float* out) {// deli: n is embedding dimension, d is head dimension, kv_head_dim, or others. 
	
	/* pure naive
	int BLOCK_SIZE = 32; // arbitrary
	dim3 blocks;
	blocks.x = (n + BLOCK_SIZE - 1)/BLOCK_SIZE;
	blocks.y = d;
	dim3 tpb;
	tpb.x = BLOCK_SIZE;
	tpb.y = 1;
	matmul<<<blocks, tpb>>>(A, x, n, d, out);
	*/

	/*partial utilization
	int MAX_THREADS_PER_BLOCK = 1024;
	matmul<<<(n + MAX_THREADS_PER_BLOCK -1)/MAX_THREADS_PER_BLOCK, MAX_THREADS_PER_BLOCK>>>(A, x, n, d, out); 
	*/

	/*partial utilization2 naive matmul: thread per row
	int MAX_THREADS_PER_BLOCK = 1024;
	int BLOCK_SIZE = WARP_SIZE;
	//needs to used "d" rather than "n", in case d > n (since the called function uses "d" as threshold) 
	matmul_full_utilization<<<(d + MAX_THREADS_PER_BLOCK -1)/MAX_THREADS_PER_BLOCK, MAX_THREADS_PER_BLOCK>>>(A, x, n, d, out); 
	*/
	/*
	int BLOCK_SIZE = WARP_SIZE;
	matmul_full_utilization<<<d, BLOCK_SIZE>>>(A, x, n, d, out);
	*/
	int BLOCK_SIZE = WARP_SIZE;
	matmul_wide<<<d, BLOCK_SIZE>>>(A, x, n, d, out);
	
	
}

template <typename T>
inline void dispatch_fused_matmul(const T* A, const float* x, int n, int d, float* out) {// deli: n is embedding dimension, d is head dimension, kv_head_dim, or others. 
	int BLOCK_SIZE = WARP_SIZE;
	fused_matmul_full_utilization<<<d, BLOCK_SIZE>>>(A, x, n, d, out);
}



template <typename T>
__global__
void fused_qkv_matmul_clip(
	const T* wq,      // (q_dim, dim)
	const T* wk,      // (kv_dim, dim)
	const T* wv,      // (kv_dim, dim)
	const float* x,   // (dim,)
	int dim,          // input dimension
	int q_dim,        // n_heads * head_dim
	int kv_dim,       // n_kv_heads * head_dim
	float qkv_clip,   // clipping value
	float* q_out,     // (q_dim,)
	float* k_out,     // (kv_dim,)
	float* v_out      // (kv_dim,)
) {
	// Each warp handles one row of either Q, K, or V output
	int warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
	int total_rows = q_dim + 2 * kv_dim;
	if (warp_id >= total_rows) return;
	
	// Determine which matrix (Q, K, or V) we're computing
	const T* w;
	float* out;
	if (warp_id < q_dim) {
		// Computing Q
		w = wq + warp_id * dim;
		out = q_out + warp_id;
	} else if (warp_id < q_dim + kv_dim) {
		// Computing K
		w = wk + (warp_id - q_dim) * dim;
		out = k_out + (warp_id - q_dim);
	} else {
		// Computing V
		w = wv + (warp_id - q_dim - kv_dim) * dim;
		out = v_out + (warp_id - q_dim - kv_dim);
	}

	// Compute matrix multiplication for this row
	// Since block is 1-dimensional, thread ID is same as threadIdx.x,
	// and warp partitions thread IDs
	int offset = threadIdx.x % warpSize;
	float row_sum = matmul_row(w, x, offset, dim);
	// Write result with clipping
	if (offset == 0) {
		row_sum = row_sum < -qkv_clip ? -qkv_clip : (row_sum > qkv_clip ? qkv_clip : row_sum);
		*out = row_sum;
	}
}

__global__
void attn(
	const float* kb,  // (max_seq_len, n_kv_heads, head_dim) 
	const float* q,   // (n_heads, head_dim)
	int head_dim, 
	int kv_len, 
	int max_seq_len, 
	int n_heads, 
  int n_kv_heads,
	float* out        // (n_heads, kv_len)
) {
	int group = blockIdx.y;
	int t = blockIdx.x * blockDim.x + threadIdx.x;
	int h = blockIdx.y * blockDim.y + threadIdx.y;
	if (t >= kv_len || h >= n_heads) return;
	
	const float* query = q + h * head_dim;
	const float* key = kb + n_kv_heads * head_dim * t + head_dim * group;
	float score = 0.0;
	for (int i = 0; i < head_dim; i++) {
		score += query[i] * key[i];
	}
  out[h * max_seq_len + t] = score / sqrtf((float)head_dim);
}

__global__
void attn_softmax(
	const float* att, 
	int seq_len, 
	int max_seq_len, 
	int n_heads, 
	float* out
) {
	int offset = threadIdx.x;
	int h = blockIdx.x;
	int block_size = blockDim.x;
	if (h >= n_heads) return;
	
	const float* atth = att + max_seq_len * h;
	float* outh = out + max_seq_len * h;
	
	float score_max = -FLT_MAX;
	for (int t = offset; t < seq_len; t += block_size) {
		if (atth[t] > score_max) {
			score_max = atth[t];
		}
	}
	score_max = block_all_reduce_max(score_max);
	float score_sum = 0.0f;
	for (int t = offset; t < seq_len; t += block_size) {
		outh[t] = expf(atth[t] - score_max);
		score_sum += outh[t];
	}
	score_sum = block_all_reduce_sum(score_sum);
	for (int t = offset; t < seq_len; t += block_size) {
		outh[t] /= score_sum;
	}
}

/*
//naive att_mix
__global__
void att_mix(
	const float* vb,  // (max_seq_len, n_kv_heads, head_dim) 
	const float* att, // (n_heads, kv_len)
	int head_dim, 
	int n_heads, 
	int n_kv_heads,
	int seq_len, 
	int max_seq_len, 
	float* out // (n_heads, head_dim)
) {
	// PRECOND: blocks are 1-D and blockDim.x == warpSize
	int h = blockIdx.x;
	int group_size = n_heads / n_kv_heads;
	int g = h / group_size;
	int i = blockIdx.y;
	int offset = threadIdx.x;
  int kv_stride = n_kv_heads * head_dim;
	
	const float* atth = att + max_seq_len * h;
	const float* vh = vb + head_dim * g;
	float* outh = out + head_dim * h;
	
	float sum = 0.0;
	for (int t = offset; t < seq_len; t += warpSize) {
		sum += vh[kv_stride * t + i] * atth[t];
	}
	sum = warp_reduce_sum(sum);
	if (offset == 0) outh[i] = sum;
}
*/
__global__
void rmsnorm(const float* x, const float* weight, int size, float eps, float* out) {
	// PRECOND: only one 1-D block is launched
	float rms = 0.0;
	int offset = threadIdx.x;
	for (int i = offset; i < size; i += blockDim.x) {
		rms += x[i] * x[i];
	}
	rms = block_all_reduce_sum(rms);
	rms = sqrtf(rms / size + eps);
	float scale = 1.0 / rms;
	for (int i = offset; i < size; i += blockDim.x) {
		out[i] = x[i] * scale * weight[i];
	}
}

__global__
void rope(
	const float* x, int d, int head_dim, int pos, float theta, int rotary_dim, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = 2 * (blockDim.x * blockIdx.x + threadIdx.x);
	if (i >= d) return;
	
	int j_head = i % head_dim;
	float freq = j_head >= rotary_dim ? 0.f : 1.0f / powf(theta, (float)j_head / (float)rotary_dim);
	float val = pos * freq;
	float fcr = cosf(val);
	float fci = sinf(val);
	
	float v0 = x[i];
	float v1 = x[i + 1];
	out[i] = v0 * fcr - v1 * fci;
	out[i + 1] = v0 * fci + v1 * fcr;
}

/*
//better att_mix
__global__
void att_mix(
  const float* vb,  // (max_seq_len, n_kv_heads, head_dim) 
  const float* att, // (n_heads, kv_len)
  int head_dim, 
  int n_heads, 
  int n_kv_heads,
  int seq_len, 
  int max_seq_len, 
  float* out // (n_heads, head_dim)
) {
  // PRECOND: blocks are 2-D (warp_size, t_stride)
  int h = blockIdx.x;
  int group_size = n_heads / n_kv_heads;
  int g = h / group_size;
  int kv_stride = n_kv_heads * head_dim;
  
  const float* atth = att + max_seq_len * h;
  const float* vh = vb + head_dim * g;
  float* outh = out + head_dim * h;
  
  int warp_id = threadIdx.y;
  int t_stride = blockDim.y;
  
  // Capacity 32 since there can be at most 32 warps in a block.
  __shared__ float shared[32];
  
  for (int i = threadIdx.x; i < head_dim; i += warpSize) {
    if (warp_id == 0) {
      shared[threadIdx.x] = 0;
    }
    __syncthreads();
    float sum = 0.0;
    for (int t = warp_id; t < seq_len; t += t_stride) {
      sum += vh[kv_stride * t + i] * atth[t];	
    }
    atomicAdd(&shared[threadIdx.x], sum);
    __syncthreads();
    if (warp_id == 0) {
      outh[i] = shared[threadIdx.x];
      shared[threadIdx.x] = 0;
    }
  }
}
*/

__global__
void att_mix(
  const half* vb,  // (max_seq_len, n_kv_heads, head_dim) 
  const float* att, // (n_heads, kv_len)
  int head_dim, 
  int n_heads, 
  int n_kv_heads,
  int seq_len, 
  int max_seq_len, 
  float* out // (n_heads, head_dim)
) {
  // PRECOND: blocks are 2-D (warp_size, t_stride)
  int h = blockIdx.x;
  int group_size = n_heads / n_kv_heads;
  int g = h / group_size;
  int kv_stride = n_kv_heads * head_dim;
  
  const float* atth = att + max_seq_len * h;
  const half* vh = vb + head_dim * g;
  float* outh = out + head_dim * h;
  
  int warp_id = threadIdx.y;
  int t_stride = blockDim.y;
  
  // Each lane of the warp accumulates across 2 head elements at a time.
  // NOTE: Assumes warpSize is 32
  __shared__ float shared0[32]; // shared0[i] == chunk[2*i]
  __shared__ float shared1[32]; // shared1[i] == chunk[2*i+1]
  
  for (int i = 2*threadIdx.x; i < head_dim; i += 2*warpSize) {
    if (warp_id == 0) {
      shared0[threadIdx.x] = 0;
      shared1[threadIdx.x] = 0;
    }
    __syncthreads();
    float2 sum01 = make_float2(0.0, 0.0);
    constexpr int UNROLL = 16;
    half2 v01_0; float att_0; 
    half2 v01_1; float att_1; 
    half2 v01_2; float att_2; 
    half2 v01_3; float att_3;
    half2 v01_4; float att_4;
    half2 v01_5; float att_5;
    half2 v01_6; float att_6;
    half2 v01_7; float att_7;
    half2 v01_8; float att_8; 
    half2 v01_9; float att_9; 
    half2 v01_10; float att_10; 
    half2 v01_11; float att_11;
    half2 v01_12; float att_12;
    half2 v01_13; float att_13;
    half2 v01_14; float att_14;
    half2 v01_15; float att_15;
    int t = warp_id;
    for (int ctr = 0; ctr < seq_len / t_stride; t += t_stride, ctr++) {
      int ctr_mod = ctr % UNROLL;
      if (ctr_mod == 0) {
        // prefetch every UNROLL iterations
        #define PREFETCH(j) \
          v01_##j = *((half2*)&vh[kv_stride * (t + j*t_stride) + i]); \
          att_##j = atth[t + j*t_stride];
        PREFETCH(0)
        PREFETCH(1)
        PREFETCH(2)
        PREFETCH(3)
        PREFETCH(4)
        PREFETCH(5)
        PREFETCH(6)
        PREFETCH(7)
        PREFETCH(8)
        PREFETCH(9)
        PREFETCH(10)
        PREFETCH(11)
        PREFETCH(12)
        PREFETCH(13)
        PREFETCH(14)
        PREFETCH(15)
        #undef PREFETCH
      }
      // pull one value out of prefetch batch
      float2 v01;
      float att_t;
      switch (ctr_mod) {
        #define CASE(j) \
          case j: v01 = __half22float2(v01_##j); att_t = att_##j; break;
        CASE(0)
        CASE(1)
        CASE(2)
        CASE(3)
        CASE(4)
        CASE(5)
        CASE(6)
        CASE(7)
        CASE(8)
        CASE(9)
        CASE(10)
        CASE(11)
        CASE(12)
        CASE(13)
        CASE(14)
        CASE(15)
        #undef CASE
      }
      // Sadly CUDA does not have float2 SIMD ops
      sum01.x += v01.x * att_t;
      sum01.y += v01.y * att_t;
    }
    for (; t < seq_len; t += t_stride) {
      float2 v01 = __half22float2(*((half2*)&vh[kv_stride * t + i]));
      float att_t = atth[t];
      // Sadly CUDA does not have float2 SIMD ops
      sum01.x += v01.x * att_t;
      sum01.y += v01.y * att_t;
    }
    atomicAdd(&shared0[threadIdx.x], sum01.x);
    atomicAdd(&shared1[threadIdx.x], sum01.y);
    __syncthreads();
    if (warp_id == 0) {
      float even = shared0[threadIdx.x];
      float odd = shared1[threadIdx.x];
      *((float2*)&outh[i]) = make_float2(even, odd);
      shared0[threadIdx.x] = 0;
      shared1[threadIdx.x] = 0;
    }
  }
}

__global__
void add_residuals(
	const float* x, const float* y, int d, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= d) return;
	
	out[i] = x[i] + y[i];
}

__global__
void clip(
	const float* x, float v, int d, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= d) return;
	
	out[i] = x[i] < -v ? -v : (x[i] > v ? v : x[i]);
}

__global__
void glu_silu(
	const float* x, const float* weight, int d, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= d) return;
	
	out[i] = weight[i] * x[i] / (1.0f + expf(-x[i]));
}

__global__
void glu_gelu(
	const float* x, const float* weight, int d, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= d) return;
	
	float v = x[i];
	out[i] =
		weight[i] * 
		0.5f * v * (1.0f + tanhf(0.797885f * (v + 0.044715f * v * v * v)));
}

// TODO: consolidate copy_embedding and copy_kv_entry into 1 memcpy kernel
__global__
void copy_embedding(
	const float* token_embedding_table, int dim, int token, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= dim) return;
	
	const float* v = token_embedding_table + dim * token;
	out[i] = v[i];
}

__global__
void copy_embedding(
	const half* token_embedding_table, int dim, int token, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= dim) return;
	
	const half* v = token_embedding_table + dim * token;
	out[i] = __half2float(v[i]);
}

__global__
void copy_kv_entry(
	const float* in, int kv_pos, int kv_dim, float* kb
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= kv_dim) return;
	
	kb[kv_pos * kv_dim + i] = in[i];
}

template <typename T>
void Block::_block_cuda(
	InferenceState& s, int pos, int kv_pos, int kv_len
) const {
	const Config& c = *_config;
	
	// attention pre-norm
	switch (c.norm_type) {
    case LayerNormType::RMSNorm: {
	    rmsnorm<<<1, max_threads_per_block>>>(
				s.x(), rms_att_weight(), c.dim, c.norm_eps, s.xb()
			);
      break;
    }
  }
  
  int q_dim = c.n_heads * c.head_dim;
  int kv_dim = c.n_kv_heads * c.head_dim;

  // qkv matmuls for this position
  dispatch_matmul<T>(wq<T>(), s.xb(), c.dim, q_dim, s.q());
  dispatch_matmul<T>(wk<T>(), s.xb(), c.dim, kv_dim, s.k());
  dispatch_matmul<T>(wv<T>(), s.xb(), c.dim, kv_dim, s.v());
  int total_rows = q_dim + 2 * kv_dim;  // Total rows across Q, K, V

  
  // some models require clipping qkv values
  clip<<<
	  (q_dim + max_threads_per_block - 1)/max_threads_per_block, 
	  max_threads_per_block
  >>>(s.q(), c.qkv_clip, q_dim, s.q());
  clip<<<
	  (kv_dim + max_threads_per_block - 1)/max_threads_per_block, 
	  max_threads_per_block
  >>>(s.k(), c.qkv_clip, kv_dim, s.k());
  clip<<<
	  (kv_dim + max_threads_per_block - 1)/max_threads_per_block,
	  max_threads_per_block
  >>>(s.v(), c.qkv_clip, kv_dim, s.v());
  
  // RoPE relative positional encoding: complex-valued rotate q and k in each head
  rope<<<
	  (q_dim + max_threads_per_block - 1)/max_threads_per_block,
	  max_threads_per_block
  >>>(
		s.q(), q_dim, c.head_dim, pos, c.rope_theta, c.rotary_dim, s.q()
	);
	rope<<<
	  (kv_dim + max_threads_per_block - 1)/max_threads_per_block,
	  max_threads_per_block
  >>>(
		s.k(), kv_dim, c.head_dim, pos, c.rope_theta, c.rotary_dim, s.k()
	);
	
	// key and value point to the kv cache
	float* kb = key_cache();
	float* vb = value_cache();
	half* kb = (half*)key_cache();
  	half* vb = (half*)value_cache();
	copy_kv_entry<<<
		(kv_dim + max_threads_per_block - 1)/max_threads_per_block, 
		max_threads_per_block
	>>>(
		s.k(), kv_pos, kv_dim, kb
	);
	copy_kv_entry<<<
		(kv_dim + max_threads_per_block - 1)/max_threads_per_block, 
		max_threads_per_block
	>>>(
		s.v(), kv_pos, kv_dim, vb
	);
	
	// multihead attention: dot products and softmax
	{
		dim3 tpb;
		tpb.x = warp_size;
		tpb.y = c.n_heads / c.n_kv_heads;
		dim3 blocks;
		blocks.x = (kv_len + tpb.x - 1) / tpb.x;
		blocks.y = (c.n_heads + tpb.y - 1) / tpb.y;
		attn<<<blocks, tpb>>>(
			kb, s.q(), c.head_dim, kv_len, c.max_seq_len, c.n_heads, c.n_kv_heads, s.att()
		);
		attn_softmax<<<c.n_heads, warp_size>>>(
			s.att(), kv_len, c.max_seq_len, c.n_heads, s.att()
		);
	}
	/*
  // multihead attention: mix values with attention scores
	{
		dim3 tpb;
		tpb.x = warp_size;
		dim3 blocks;
		blocks.x = c.n_heads;
		blocks.y = c.head_dim;
		att_mix<<<blocks, tpb>>>(
			vb, s.att(),
			c.head_dim, c.n_heads, c.n_kv_heads, 
			kv_len, c.max_seq_len, s.xb2()
		);
	}
	*/
	{
		dim3 tpb;
		tpb.x = warp_size;
		tpb.y = min(kv_len, max_threads_per_block / warp_size);
		dim3 blocks;
		blocks.x = c.n_heads;
		att_mix<<<blocks, tpb>>>(
		vb, s.att(),
		c.head_dim, c.n_heads, c.n_kv_heads, 
		kv_len, c.max_seq_len, s.xb2());
	}
	// final matmul projection via wo, using `hb` as temp storage
	dispatch_matmul<T>(wo<T>(), s.xb2(), q_dim, c.dim, s.hb());
	//dispatch_fused_matmul<T>(wo<T>(), s.xb2(), q_dim, c.dim, s.x());
	
	// attn residual back into x
	add_residuals<<<
		(c.dim + max_threads_per_block - 1)/max_threads_per_block, 
		max_threads_per_block
	>>>(
		s.x(), s.hb(), c.dim, s.x()
	);
	
	
	// ffn pre-norm
	switch (c.norm_type) {
    case LayerNormType::RMSNorm: {
	    rmsnorm<<<1, max_threads_per_block>>>(
				s.x(), rms_ffn_weight(), c.dim, c.norm_eps, s.xb()
			);
      break;
    }
  }
	
	// mix self.w2(F.silu(self.w1(x)) * self.w3(x))
  // Note this is a feedforward with a GLU, not a simple MLP.
  dispatch_matmul<T>(w1<T>(), s.xb(), c.dim, c.hidden_dim, s.hb());
  dispatch_matmul<T>(w3<T>(), s.xb(), c.dim, c.hidden_dim, s.hb2());
  switch (c.act) {
	  case ActivationType::GELU: {
		  glu_gelu<<<
			  (c.hidden_dim + max_threads_per_block - 1)/max_threads_per_block, 
			  max_threads_per_block
		  >>>(
				s.hb(), s.hb2(), c.hidden_dim, s.hb()
			);
		  break;
	  }
	  case ActivationType::SILU: {
		  glu_silu<<<
			  (c.hidden_dim + max_threads_per_block - 1)/max_threads_per_block, 
			  max_threads_per_block
		  >>>(
				s.hb(), s.hb2(), c.hidden_dim, s.hb()
			);
		  break;
	  }
  }
    //dispatch_fused_matmul<T>(w2<T>(), s.hb(), c.hidden_dim, c.dim, s.x());
  	dispatch_matmul<T>(w2<T>(), s.hb(), c.hidden_dim, c.dim, s.xb2());
  
	// ffn residual back into x
	add_residuals<<<
		(c.dim + max_threads_per_block - 1)/max_threads_per_block,
		max_threads_per_block
	>>>(
		s.x(), s.xb2(), c.dim, s.x()
	);
	
}

void mha_cuda(
  float* xout,  // (n_heads, head_dim)
  float* att,   // (n_heads, max_seq_len)
  float* kb,    // (max_seq_len, n_kv_heads, head_dim)
  float* vb,    // (max_seq_len, n_kv_heads, head_dim)
  float* q,     // (n_heads, head_dim)
  int head_dim, int kv_len, int max_seq_len, int n_heads, int n_kv_heads
) {
  int warp_size = 32;
  // all cuda uploads leak forever...
  register_cuda_host(xout, n_heads * head_dim * sizeof(float));
  register_cuda_host(att, n_heads * max_seq_len * sizeof(float));
  kb = static_cast<float*>(upload_cuda(kb, max_seq_len * n_kv_heads * head_dim * sizeof(float)));
  vb = static_cast<float*>(upload_cuda(vb, max_seq_len * n_kv_heads * head_dim * sizeof(float)));
  q = static_cast<float*>(upload_cuda(q, n_heads * head_dim * sizeof(float)));
  // multihead attention: dot products and softmax
	{
		dim3 tpb;
		tpb.x = warp_size;
		tpb.y = n_heads / n_kv_heads;
		dim3 blocks;
		blocks.x = (kv_len + tpb.x - 1) / tpb.x;
		blocks.y = (n_heads + tpb.y - 1) / tpb.y;
		attn<<<blocks, tpb>>>(
			kb, q, head_dim, kv_len, max_seq_len, n_heads, n_kv_heads, att
		);
		attn_softmax<<<n_heads, warp_size>>>(
			att, kv_len, max_seq_len, n_heads, att
		);
	}
  // multihead attention: mix values with attention scores
	{
		dim3 tpb;
		tpb.x = warp_size;
		dim3 blocks;
		blocks.x = n_heads;
		blocks.y = head_dim;
		att_mix<<<blocks, tpb>>>(
			vb, att,
			head_dim, n_heads, n_kv_heads, 
			kv_len, max_seq_len, xout
		);
	}
  CUDA_CHECK(hipDeviceSynchronize()); // After this, xout contains output
	CUDA_CHECK(hipGetLastError()); // check for kernel launch errors
  unregister_cuda_host(xout);
  unregister_cuda_host(att);
}

void matmul_cuda(float* xout, float* x, float* w, int n, int d) {
  int warp_size = 32;
  int max_threads_per_block = 1024;
  // A (d,n) @ x (n,) -> out (d,)

  // all cuda uploads leak forever...
  register_cuda_host(xout, d * sizeof(float));
  x = static_cast<float*>(upload_cuda(x, n * sizeof(float)));
  w = static_cast<float*>(upload_cuda(w, n * d * sizeof(float)));
  dispatch_matmul<float>(w, x, n, d, xout);
  CUDA_CHECK(hipDeviceSynchronize()); // After this, xout contains output
	CUDA_CHECK(hipGetLastError()); // check for kernel launch errors
  unregister_cuda_host(xout);
}

void ffn_cuda(
  float* xout, float* x, 
  float* w1, float* w2, float* w3, 
  int hidden_dim, int dim,
  ActivationType act
) {
  int warp_size = 32;
  int max_threads_per_block = 1024;
  // all cuda uploads leak forever...
  register_cuda_host(xout, dim * sizeof(float));
  x = static_cast<float*>(upload_cuda(x, dim * sizeof(float)));
  w1 = static_cast<float*>(upload_cuda(w1, hidden_dim * dim * sizeof(float)));
  w2 = static_cast<float*>(upload_cuda(w2, dim * hidden_dim * sizeof(float)));
  w3 = static_cast<float*>(upload_cuda(w3, hidden_dim * dim * sizeof(float)));
  float* hb = new float[hidden_dim];
  float* hb2 = new float[hidden_dim];
  hb = static_cast<float*>(upload_cuda(hb, hidden_dim * sizeof(float)));
  hb2 = static_cast<float*>(upload_cuda(hb2, hidden_dim * sizeof(float)));
  // hb, hb2 leak forever on cpu too...

  // mix self.w2(F.silu(self.w1(x)) * self.w3(x))
  // Note this is a feedforward with a GLU, not a simple MLP.
  dispatch_matmul<float>(w1, x, dim, hidden_dim, hb);
  dispatch_matmul<float>(w3, x, dim, hidden_dim, hb2);
  switch (act) {
	  case ActivationType::GELU: {
		  glu_gelu<<<
			  (hidden_dim + max_threads_per_block - 1)/max_threads_per_block, 
			  max_threads_per_block
		  >>>(
				hb, hb2, hidden_dim, hb
			);
		  break;
	  }
	  case ActivationType::SILU: {
		  glu_silu<<<
			  (hidden_dim + max_threads_per_block - 1)/max_threads_per_block, 
			  max_threads_per_block
		  >>>(
				hb, hb2, hidden_dim, hb
			);
		  break;
	  }
  }
  
  dispatch_matmul<float>(w2, hb, hidden_dim, dim, xout);
  CUDA_CHECK(hipDeviceSynchronize()); // After this, xout contains output
	CUDA_CHECK(hipGetLastError()); // check for kernel launch errors
  unregister_cuda_host(xout);
}

template void Block::_block_cuda<float>(InferenceState&, int, int, int) const;
template void Block::_block_cuda<half>(InferenceState&, int, int, int) const;
template<> void Block::_block_cuda<f16_t>(InferenceState& s, int pos, int kv_pos, int kv_len) const {
  _block_cuda<half>(s, pos, kv_pos, kv_len);
}

void Model::_forward_cuda(InferenceState& s, int token, int pos, InferenceMode mode) {
	const Config& c = *config;
	
  switch (c.weight_dtype) {
    case DType::F32: {
	    copy_embedding<<<
        (c.dim + max_threads_per_block - 1)/max_threads_per_block,
        max_threads_per_block
      >>>(
        static_cast<float*>(token_embedding_table), c.dim, token, s.x()
      );
      break;
    }
    case DType::F16: {
	    copy_embedding<<<
        (c.dim + max_threads_per_block - 1)/max_threads_per_block,
        max_threads_per_block
      >>>(
        static_cast<half*>(token_embedding_table), c.dim, token, s.x()
      );
      break;
    }
    default: {
      assert(false && "unsupported weight dtype for CUDA");
    }
  }
	
	// TODO: attention sinks
	int kv_pos = pos % c.max_seq_len;
	int kv_len = pos >= c.max_seq_len ? c.max_seq_len : pos + 1;
	
	// forward all layers in order
	for (auto b : blocks) {
		b->block(s, pos, kv_pos, kv_len);
	}

  if (mode == InferenceMode::HYDRATE_KV_CACHE) {
    // only hydrate the KV cache and don't compute output logits
	  CUDA_CHECK(hipGetLastError()); // check for kernel launch errors
    return;
  }
	
	// final layer norm
	switch (c.norm_type) {
		case LayerNormType::RMSNorm: {
			rmsnorm<<<1, max_threads_per_block>>>(
				s.x(), rms_final_weight, c.dim, c.norm_eps, s.x()
			);
			break;
		}
	}
	
	// classifier into logits
	switch (c.weight_dtype) {
    case DType::F32: {
	    dispatch_matmul<float>(
        static_cast<float*>(wcls), s.x(), c.dim, c.vocab_size, s.logits()
      );
      break;
    }
    case DType::F16: {
	    dispatch_matmul<half>(
        static_cast<half*>(wcls), s.x(), c.dim, c.vocab_size, s.logits()
      );
      break;
    }
    default: {
      assert(false && "unsupported weight dtype for CUDA");
    }
  }
	
	CUDA_CHECK(hipDeviceSynchronize()); // After this, s.logits contains logits of output token
	CUDA_CHECK(hipGetLastError()); // check for kernel launch errors
}